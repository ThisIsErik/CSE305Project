#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <chrono>
#include <math.h>
#include "CUDAlign.cuh"
#include <hip/hip_runtime.h>
#include "utils/types.h"

__device__ __host__ inline int get_index(int i, int j, int n) {
    return i * (n + 1) + j;
}

__global__
void CUDAlignAux(int* dp, 
    const char* A,
    const char* B,
    int external_diagonal, 
    int mi, int ma, int g, 
    int rows, int columns, 
    int BIG_ROWS, int BIG_COLUMNS,
    int m, int n,
    int alpha,
    int* max_val,
    long long* max_pos) {
    if (blockIdx.x <= external_diagonal && external_diagonal - blockIdx.x < BIG_ROWS){
        // in big grid, this block will treat the big cell indexed G_(external_diagonal-blockIdx, blockIdx)
        int topLeft_i = (external_diagonal-blockIdx.x)*rows;
        int topLeft_j = blockIdx.x * columns;
        for (int internal_diagonal = 0; internal_diagonal < rows/alpha + columns - 1; internal_diagonal++){
            // inside the big gird cell, we have rows x columns individual cells
            // our thread will take care of lines threadIdx*alpha to (threadIdx+1)*alpha - 1

            // on this internal diagonal, the current thread will take care of alpha rows on this column
            int row_thread = threadIdx.x * alpha;
            int column_thread = internal_diagonal - threadIdx.x; 
            if (column_thread >= 0 && column_thread < columns){
                // check column index is valid

                for (int offset = 0; offset < alpha; offset++){
                    // offset will say which one out of the alpha rows we process
                    int i = topLeft_i + row_thread + offset; // i = topLeft_i index of big gridcell + which rows these thread processes + offset
                    int j = topLeft_j + column_thread; // j = topLeft_j index of big gridcell + which column this thread processes for this antidiagonal

                    int p = (A[i] == B[j]) ? ma : mi;
                    int val = max(max(max(
                    dp[get_index(i, j, n)] + p,
                    dp[get_index(i + 1, j, n)] + g),
                    dp[get_index(i, j + 1, n)] + g),
                    0);
                    dp[get_index(i + 1, j + 1, n)] = val;

                    int old_max = atomicMax(max_val, val);
                    long long pos_code = (static_cast<long long>(i + 1) << 32) | (j + 1);

                    if (val > old_max) {
                        *max_pos = pos_code;
                    }
                    else if (val == old_max) {
                        atomicMax(reinterpret_cast<unsigned long long*>(max_pos),
                                static_cast<unsigned long long>(pos_code));
                    }
                    
                }
            }
            __syncthreads(); // sync all threads within the block before moving on to the next internal diagonal
        }
    }
}

SWResultScore CUDAlign(
    const std::string& A,
    const std::string& B,
    int mi, int ma, int g
) {
    const size_t BLOCKS_NUM = 32;
    const size_t THREADS_PER_BLOCK = 32;
    const size_t ROWS_PER_THREAD = 4;

    const int m = static_cast<int>(A.size());
    const int n = static_cast<int>(B.size());

    int host_max_val = 0;
    long long host_max_pos = 0;

    int* dev_max_val;
    long long* dev_max_pos;
    hipMalloc(&dev_max_val, sizeof(int));
    hipMalloc(&dev_max_pos, sizeof(long long));
    hipMemcpy(dev_max_val, &host_max_val, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_max_pos, &host_max_pos, sizeof(long long), hipMemcpyHostToDevice);

    const size_t BIG_COLUMNS = BLOCKS_NUM; // B
    const size_t BIG_ROWS = m/(THREADS_PER_BLOCK * ROWS_PER_THREAD); // m/(alpha*T)
    const size_t columns = n/BLOCKS_NUM; // C = n/B
    const size_t rows = THREADS_PER_BLOCK * ROWS_PER_THREAD; // R = alpha*T
    const size_t alpha = ROWS_PER_THREAD; // alpha

    // moving the data to device 
    int* dpd;
    hipMalloc(&dpd, (m + 1) * (n + 1) * sizeof(int));
    hipMemset(dpd, 0, (m + 1) * (n + 1) * sizeof(int));

    char* A_dev;
    char* B_dev;
    hipMalloc(&A_dev, m * sizeof(char));
    hipMalloc(&B_dev, n * sizeof(char));
    hipMemcpy(A_dev, A.c_str(), m * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(B_dev, B.c_str(), n * sizeof(char), hipMemcpyHostToDevice);

    // computing on GPU
    for (int external_diagonal = 0; external_diagonal < BIG_COLUMNS + BIG_ROWS - 1; external_diagonal++) {
        CUDAlignAux<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>(
            dpd,
            A_dev, B_dev,
            external_diagonal,
            mi, ma, g,
            rows, columns,
            BIG_ROWS, BIG_COLUMNS,
            m, n,
            alpha,
            dev_max_val,
            dev_max_pos
        );
    }

    // copying the result back
    hipMemcpy(&host_max_val, dev_max_val, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&host_max_pos, dev_max_pos, sizeof(long long), hipMemcpyDeviceToHost);
    int max_i = static_cast<int>(host_max_pos >> 32);
    int max_j = static_cast<int>(host_max_pos & 0xFFFFFFFF);
    // std::cout << "Max value: " << host_max_val << " at (" << max_i << ", " << max_j << ")\n";

    hipFree(dpd);
    hipFree(A_dev);
    hipFree(B_dev);
    hipFree(dev_max_val);
    hipFree(dev_max_pos);

    return {host_max_val, max_i, max_j};
}
