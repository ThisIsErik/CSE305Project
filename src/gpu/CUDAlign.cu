#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <chrono>
#include <math.h>
#include "CUDAlign.cuh"
#include <hip/hip_runtime.h>

__device__ __host__ inline int get_index(int i, int j, int n) {
    return i * (n + 1) + j;
}

__global__
void CUDAlignAux(int* dp, 
    const char* A,
    const char* B,
    int external_diagonal, 
    int mi, int ma, int g, 
    int rows, int columns, 
    int BIG_ROWS, int BIG_COLUMNS,
    int m, int n,
    int alpha) {
    if (blockIdx.x <= external_diagonal && external_diagonal - blockIdx.x < BIG_ROWS){
        // in big grid, this block will treat the big cell indexed G_(external_diagonal-blockIdx, blockIdx)
        int topLeft_i = (external_diagonal-blockIdx.x)*rows;
        int topLeft_j = blockIdx.x * columns;
        for (int internal_diagonal = 0; internal_diagonal < rows/alpha + columns - 1; internal_diagonal++){
            // inside the big gird cell, we have rows x columns individual cells
            // our thread will take care of lines threadIdx*alpha to (threadIdx+1)*alpha - 1

            // on this internal diagonal, the current thread will take care of alpha rows on this column
            int row_thread = threadIdx.x * alpha;
            int column_thread = internal_diagonal - threadIdx.x; 
            if (column_thread >= 0 && column_thread < columns){
                // check column index is valid

                for (int offset = 0; offset < alpha; offset++){
                    // offset will say which one out of the alpha rows we process
                    int i = topLeft_i + row_thread + offset; // i = topLeft_i index of big gridcell + which rows these thread processes + offset
                    int j = topLeft_j + column_thread; // j = topLeft_j index of big gridcell + which column this thread processes for this antidiagonal

                    int p = (A[i] == B[j]) ? ma : mi;
                    int val = max(max(max(
                    dp[get_index(i, j, n)] + p,
                    dp[get_index(i + 1, j, n)] + g),
                    dp[get_index(i, j + 1, n)] + g),
                    0);
                    dp[get_index(i + 1, j + 1, n)] = val;
                }
            }
            __syncthreads(); // sync all threads within the block before moving on to the next internal diagonal
        }
    }
}

void CUDAlign(
    const std::string& A,
    const std::string& B,
    int mi, int ma, int g
) {
    const size_t BLOCKS_NUM = 32;
    const size_t THREADS_PER_BLOCK = 32;
    const size_t ROWS_PER_THREAD = 4;

    const int m = static_cast<int>(A.size());
    const int n = static_cast<int>(B.size());
    int* dp = new int[(m + 1) * (n + 1)]();

    // int max_val = 0;
    // std::pair<int, int> max_pos = {0, 0};

    const size_t BIG_COLUMNS = BLOCKS_NUM; // B
    const size_t BIG_ROWS = m/(THREADS_PER_BLOCK * ROWS_PER_THREAD); // m/(alpha*T)
    const size_t columns = n/BLOCKS_NUM; // C = n/B
    const size_t rows = THREADS_PER_BLOCK * ROWS_PER_THREAD; // R = alpha*T
    const size_t alpha = ROWS_PER_THREAD; // alpha

    // moving the data to device 
    int* dpd;
    hipMalloc(&dpd, (m + 1) * (n + 1) * sizeof(int));
    hipMemcpy(dpd, dp, (m + 1) * (n + 1) * sizeof(int), hipMemcpyHostToDevice);

    char* A_dev;
    char* B_dev;
    hipMalloc(&A_dev, m * sizeof(char));
    hipMalloc(&B_dev, n * sizeof(char));
    hipMemcpy(A_dev, A.c_str(), m * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(B_dev, B.c_str(), n * sizeof(char), hipMemcpyHostToDevice);

    // computing on GPU
    for (int external_diagonal = 0; external_diagonal < BIG_COLUMNS + BIG_ROWS - 1; external_diagonal++) {
        CUDAlignAux<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>(
            dpd,
            A_dev, B_dev,
            external_diagonal,
            mi, ma, g,
            rows, columns,
            BIG_ROWS, BIG_COLUMNS,
            m, n,
            alpha
        );
    }

    // copying the result back
    hipMemcpy(dp, dpd, (m + 1) * (n + 1) * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dpd);
    hipFree(A_dev);
    hipFree(B_dev);
}

// //----------------------------------------------------

// int main(int argc, char* argv[]) {
//     // setting the random seed to get the same result each time
//     srand(42);

//     // taking as input, which algo to run
//     int alg_ind = std::stoi(argv[1]);

//     // Generating data
//     size_t N = 1 << 27;
//     double* x = (double*) malloc(N * sizeof(double));
//     double* y = (double*) malloc(N * sizeof(double));
//     for (size_t i = 0; i < N; ++i) {
//           x[i] = static_cast <double> (rand()) / static_cast <double> (RAND_MAX);
//           y[i] = static_cast <double> (rand()) / static_cast <double> (RAND_MAX);
//     }
 
//     // Warming up the kernel
//     auto start = std::chrono::steady_clock::now();
//     warmup<<<1,1>>>();
//     auto finish = std::chrono::steady_clock::now();
//     auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(finish - start).count(); 
//     std::cout << "Warm-up time: " << elapsed << std::endl;
 

//     // Allocating the result
//     double* result = (double*) malloc(N * sizeof(double));
//     start = std::chrono::steady_clock::now();
//     switch (alg_ind) {
//         case 0: 
//             Add(x, y, result, N);
//             break;
//         case 1:
//             AddGPU(x, y, result, N);
//             break;
//         case 2:
//             AddGPU2(x, y, result, N);
//             break;
//     }
//     finish = std::chrono::steady_clock::now();
//     elapsed = std::chrono::duration_cast<std::chrono::microseconds>(finish - start).count(); 
//     std::cout << "Elapsed time: " << elapsed << std::endl;
    
//     delete[] x;
//     delete[] y;
//     delete[] result;
//     return 0;
// }
