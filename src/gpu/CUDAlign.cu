#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <chrono>
#include <math.h>
#include "CUDAlign.cuh"
#include <hip/hip_runtime.h>


__global__
void CUDAlignAux(int* dp, int external_diagonal, int mi, int ma, int g) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
}

void CUDAlign(
    const std::string& A,
    const std::string& B,
    int mi, int ma, int g
) {
    const size_t BLOCKS_NUM = 32;
    const size_t THREADS_PER_BLOCK = 32;
    const size_t ROWS_PER_THREAD = 4;

    const int m = A.size();
    const int n = B.size();
    int* dp = new int[(m + 1) * (n + 1)]();  

    int max_val = 0;
    std::pair<int, int> max_pos = {0, 0};

    const size_t BIG_COLUMNS = BLOCKS_NUM;
    const size_t BIG_ROWS = m/(THREADS_PER_BLOCK * ROWS_PER_THREAD);

    // moving the data to device 
    int* dpd;
    hipMalloc(&dpd, (m + 1) * (n + 1) * sizeof(int));
    hipMemcpy(dpd, dp, (m + 1) * (n + 1) * sizeof(int), hipMemcpyHostToDevice);

    // computing on GPU
    for (int external_diagonal=0;external_diagonal<BIG_COLUMNS + BIG_ROWS - 1; external_diagonal++){
        CUDAlignAux<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>(dpd, external_diagonal, mi, ma, g);
    }

    // copying the result back
    hipMemcpy(dpd, dp, (m + 1) * (n + 1) * sizeof(int), hipMemcpyDeviceToHost);
  
    // Free memory
    hipFree(dpd);
}

// //----------------------------------------------------

// int main(int argc, char* argv[]) {
//     // setting the random seed to get the same result each time
//     srand(42);

//     // taking as input, which algo to run
//     int alg_ind = std::stoi(argv[1]);

//     // Generating data
//     size_t N = 1 << 27;
//     double* x = (double*) malloc(N * sizeof(double));
//     double* y = (double*) malloc(N * sizeof(double));
//     for (size_t i = 0; i < N; ++i) {
//           x[i] = static_cast <double> (rand()) / static_cast <double> (RAND_MAX);
//           y[i] = static_cast <double> (rand()) / static_cast <double> (RAND_MAX);
//     }
 
//     // Warming up the kernel
//     auto start = std::chrono::steady_clock::now();
//     warmup<<<1,1>>>();
//     auto finish = std::chrono::steady_clock::now();
//     auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(finish - start).count(); 
//     std::cout << "Warm-up time: " << elapsed << std::endl;
 

//     // Allocating the result
//     double* result = (double*) malloc(N * sizeof(double));
//     start = std::chrono::steady_clock::now();
//     switch (alg_ind) {
//         case 0: 
//             Add(x, y, result, N);
//             break;
//         case 1:
//             AddGPU(x, y, result, N);
//             break;
//         case 2:
//             AddGPU2(x, y, result, N);
//             break;
//     }
//     finish = std::chrono::steady_clock::now();
//     elapsed = std::chrono::duration_cast<std::chrono::microseconds>(finish - start).count(); 
//     std::cout << "Elapsed time: " << elapsed << std::endl;
    
//     delete[] x;
//     delete[] y;
//     delete[] result;
//     return 0;
// }
