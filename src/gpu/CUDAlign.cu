#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <chrono>
#include <math.h>
#include "CUDAlign.cuh"
#include <hip/hip_runtime.h>
#include "utils/types.h"

__device__ __host__ inline size_t get_index(size_t i, size_t j, size_t n) {
    return i * (n + 1ULL) + j;
}

__global__
void CUDAlignAux(unsigned short* dp, 
    const char* A,
    const char* B,
    int external_diagonal, 
    int mi, int ma, int g, 
    int rows, int columns, 
    int BIG_ROWS, int BIG_COLUMNS,
    int m, int n,
    int alpha,
    int* max_val,
    long long* max_pos) {

    if (blockIdx.x <= external_diagonal && external_diagonal - blockIdx.x < BIG_ROWS){
        int topLeft_i = (external_diagonal - blockIdx.x) * rows;
        int topLeft_j = blockIdx.x * columns;

        for (int internal_diagonal = 0; internal_diagonal < rows / alpha + columns - 1; internal_diagonal++) {
            int row_thread = threadIdx.x * alpha;
            int column_thread = internal_diagonal - threadIdx.x;

            if (column_thread >= 0 && column_thread < columns) {
                for (int offset = 0; offset < alpha; offset++) {
                    int i = topLeft_i + row_thread + offset;
                    int j = topLeft_j + column_thread;

                    if (i < 0 || j < 0 || i >= m || j >= n || i + 1 > m || j + 1 > n) {
                        printf("[OOB] Thread (%d,%d): i=%d j=%d (m=%d, n=%d)\n", blockIdx.x, threadIdx.x, i, j, m, n);
                        continue;
                    }

                    int p = (A[i] == B[j]) ? ma : mi;

                    size_t write_index = ((size_t)(i + 1)) * (n + 1ULL) + (j + 1);
                    if (write_index >= (size_t)(m + 1) * (n + 1)) {
                        printf("[ERROR] Invalid dp index at (%d,%d): index=%zu\n", i + 1, j + 1, write_index);
                        continue;
                    }

                    if (i == 0 && j == 0 && threadIdx.x == 0 && blockIdx.x == 0) {
                        printf("[TRACE] Kernel start: A[0]=%c B[0]=%c n=%d\n", A[0], B[0], n);
                    }

                    unsigned short val = max(max(max(
                        (int)dp[get_index(i, j, n)] + p,
                        (int)dp[get_index(i + 1, j, n)] + g),
                        (int)dp[get_index(i, j + 1, n)] + g),
                        0);

                    dp[write_index] = val;

                    int old_max = atomicMax(max_val, static_cast<int>(val));
                    long long pos_code = (static_cast<long long>(i + 1) << 32) | (j + 1);

                    if (val > old_max) {
                        *max_pos = pos_code;
                    } else if (val == old_max) {
                        atomicMax(reinterpret_cast<unsigned long long*>(max_pos),
                                  static_cast<unsigned long long>(pos_code));
                    }
                }
            }
            __syncthreads();
        }
    }
}

SWResultScore CUDAlign(const std::string& A, const std::string& B, int mi, int ma, int g) {
    const size_t BLOCKS_NUM = 32;
    const size_t THREADS_PER_BLOCK = 32;
    const size_t ROWS_PER_THREAD = 4;

    const int m = static_cast<int>(A.size());
    const int n = static_cast<int>(B.size());

    int host_max_val = 0;
    long long host_max_pos = 0;

    int* dev_max_val;
    long long* dev_max_pos;
    hipMalloc(&dev_max_val, sizeof(int));
    hipMalloc(&dev_max_pos, sizeof(long long));
    hipMemcpy(dev_max_val, &host_max_val, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_max_pos, &host_max_pos, sizeof(long long), hipMemcpyHostToDevice);

    const size_t rows = THREADS_PER_BLOCK * ROWS_PER_THREAD;
    const size_t columns = (n + BLOCKS_NUM - 1) / BLOCKS_NUM;
    const size_t BIG_COLUMNS = BLOCKS_NUM;
    const size_t BIG_ROWS = (m + rows - 1) / rows;
    const size_t alpha = ROWS_PER_THREAD;

    unsigned short* dpd;
    hipError_t err = hipMalloc(&dpd, (m + 1) * (n + 1) * sizeof(unsigned short));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed: " << hipGetErrorString(err) << std::endl;
        return {0, 0, 0};
    }
    hipMemset(dpd, 0, (m + 1) * (n + 1) * sizeof(unsigned short));

    char* A_dev;
    char* B_dev;
    hipMalloc(&A_dev, m * sizeof(char));
    hipMalloc(&B_dev, n * sizeof(char));
    hipMemcpy(A_dev, A.c_str(), m * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(B_dev, B.c_str(), n * sizeof(char), hipMemcpyHostToDevice);

    for (int external_diagonal = 0; external_diagonal < BIG_COLUMNS + BIG_ROWS - 1; external_diagonal++) {
        CUDAlignAux<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>(
            dpd, A_dev, B_dev, external_diagonal,
            mi, ma, g,
            rows, columns,
            BIG_ROWS, BIG_COLUMNS,
            m, n, alpha,
            dev_max_val, dev_max_pos);

        hipError_t kernel_err = hipGetLastError();
        if (kernel_err != hipSuccess) {
            std::cerr << "Kernel launch failed: " << hipGetErrorString(kernel_err) << std::endl;
            break;
        }
        hipDeviceSynchronize();
    }

    hipMemcpy(&host_max_val, dev_max_val, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&host_max_pos, dev_max_pos, sizeof(long long), hipMemcpyDeviceToHost);
    int max_i = static_cast<int>(host_max_pos >> 32);
    int max_j = static_cast<int>(host_max_pos & 0xFFFFFFFF);

    hipFree(dpd);
    hipFree(A_dev);
    hipFree(B_dev);
    hipFree(dev_max_val);
    hipFree(dev_max_pos);

    return {host_max_val, max_i, max_j};
}
